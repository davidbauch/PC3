#include "hip/hip_runtime.h"
#ifndef USECPU
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <thrust/pair.h>
#include <thrust/device_ptr.h>
#else
#include <ranges>
#include <algorithm>
#endif
#include "cuda/cuda_complex.cuh"
#include "misc/helperfunctions.hpp"

std::tuple<real_number, real_number> PC3::CUDA::minmax( complex_number* buffer, int size, bool device_pointer ) {
    #ifndef USECPU
    if ( device_pointer ) {
        thrust::device_ptr<complex_number> dev_buffer = thrust::device_pointer_cast( buffer );
        auto mm = thrust::minmax_element( thrust::device, dev_buffer, dev_buffer + size, compare_complex_abs2() );
        complex_number min = *mm.first;
        complex_number max = *mm.second;
        return std::make_tuple( CUDA::sqrt(min.x * min.x + min.y * min.y), CUDA::sqrt(max.x * max.x + max.y * max.y) );
    }
    const auto [first, second] = thrust::minmax_element( buffer, buffer + size, compare_complex_abs2() );
    #else
    const auto [first, second] = std::ranges::minmax_element( buffer, buffer + size, compare_complex_abs2() );
    #endif
    return std::make_tuple( CUDA::sqrt(CUDA::real( *first ) * CUDA::real( *first ) + CUDA::imag( *first ) * CUDA::imag( *first )), CUDA::sqrt(CUDA::real( *second ) * CUDA::real( *second ) + CUDA::imag( *second ) * CUDA::imag( *second )) );
}
std::tuple<real_number, real_number> PC3::CUDA::minmax( real_number* buffer, int size, bool device_pointer ) {
    #ifndef USECPU
    if (device_pointer) {
        thrust::device_ptr<real_number> dev_buffer = thrust::device_pointer_cast(buffer);
        auto mm = thrust::minmax_element( thrust::device, dev_buffer, dev_buffer + size, thrust::less<real_number>() );
        real_number min = *mm.first;
        real_number max = *mm.second;
        return std::make_tuple( min, max );
    }
    const auto [first, second] = thrust::minmax_element( buffer, buffer + size, thrust::less<real_number>() );
    #else
    const auto [first, second] = std::ranges::minmax_element( buffer, buffer + size, std::less<real_number>() );
    #endif
    return std::make_tuple( *first, *second );
}